#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

// Define block size
#define BLOCK_SIZE 32

// Helper macro for ceiling division
#define CEIL_DIV(M, N) (((M) + (N) - 1) / (N))

__global__ void sgemm_shared(int M, int N, int K, float alpha, const float *A, 
                             const float *B, float beta, float *C) {
    // Shared memory arrays for blocks of A and B
    __shared__ float As[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE * BLOCK_SIZE];

    // Thread identifiers
    int threadRow = threadIdx.y;  // Row within the block
    int threadCol = threadIdx.x;  // Column within the block

    // Block identifiers
    int cRow = blockIdx.y;  // Row of the block in C
    int cCol = blockIdx.x;  // Column of the block in C

    // Advance pointers to starting positions
    const float *A_start = A + cRow * BLOCK_SIZE * K;
    const float *B_start = B + cCol * BLOCK_SIZE;
    float *C_start = C + cRow * BLOCK_SIZE * N + cCol * BLOCK_SIZE;

    // Initialize the result for this thread
    float tmp = 0.0f;

    // Loop over all chunks of A and B
    for (int bkIdx = 0; bkIdx < K; bkIdx += BLOCK_SIZE) {
        // Each thread loads its corresponding elements from A and B into shared memory
        As[threadRow * BLOCK_SIZE + threadCol] = 
            A_start[threadRow * K + (threadCol + bkIdx)];
        Bs[threadRow * BLOCK_SIZE + threadCol] = 
            B_start[(threadRow + bkIdx) * N + threadCol];

        // Synchronize to make sure shared memory is fully populated
        __syncthreads();

        // Perform dot product
        for (int dotIdx = 0; dotIdx < BLOCK_SIZE; ++dotIdx) {
            tmp += As[threadRow * BLOCK_SIZE + dotIdx] * 
                   Bs[dotIdx * BLOCK_SIZE + threadCol];
        }

        // Synchronize to prevent overwriting shared memory prematurely
        __syncthreads();
    }

    // Write the computed value to the C matrix
    C_start[threadRow * N + threadCol] = alpha * tmp + 
                                         beta * C_start[threadRow * N + threadCol];
}

// Initialize matrix with random values
void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}

// Function to measure execution time
double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main() {
    // Define matrix dimensions
    const int M = 4092;  // rows of A and C
    const int N = 4092;  // columns of B and C
    const int K = 4092;  // columns of A and rows of B

    // Define SGEMM parameters
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Allocate host memory
    float *h_A = (float*)malloc(M * K * sizeof(float));
    float *h_B = (float*)malloc(K * N * sizeof(float));
    float *h_C = (float*)malloc(M * N * sizeof(float));

    // Initialize matrices with random values
    srand(time(NULL));
    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);
    init_matrix(h_C, M, N);  // Initialize C even though beta is 0 in this example

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    dim3 gridDim(CEIL_DIV(M, BLOCK_SIZE), CEIL_DIV(N, BLOCK_SIZE), 1);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);

    // Benchmark kernel execution
    printf("Running SGEMM with matrix dimensions M=%d, N=%d, K=%d\n", M, N, K);
    printf("Grid dimensions: %d x %d\n", gridDim.x, gridDim.y);
    
    double start_time = get_time();
    
    // Launch kernel
    sgemm_shared<<<gridDim, blockDim>>>(M, N, K, alpha, d_A, d_B, beta, d_C);
    
    // Synchronize and check timing
    hipDeviceSynchronize();
    double end_time = get_time();
    
    printf("Kernel execution time: %f ms\n", (end_time - start_time) * 1000.0);

    // Copy result back to host
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // Calculate and print GFLOPS
    double gflops = (2.0 * M * N * K) / ((end_time - start_time) * 1e9);
    printf("Performance: %.2f GFLOPS\n", gflops);

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}